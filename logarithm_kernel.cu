#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void logarithm(float *device_parts, float n, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < N) {
        float x = 2*idx + 1;
        device_parts[idx] = (1/x)*pow( (n - 1)/(n + 1), x);
    }
}

int num_terms = 32;
size_t size_terms = num_terms*sizeof(int);

int main() {
   int z = 16;
   float *device_parts;
   float *host_parts;
   hipMalloc((void**) &device_parts, size_terms);
   hipHostMalloc((void**) &host_parts, size_terms, hipHostMallocDefault);

   logarithm <<< 2, 16, 1 >>> (device_parts, z, num_terms);

   hipMemcpy(host_parts, device_parts, size_terms, hipMemcpyDeviceToHost);
   int i;
   float total = 0;
   for(i = 0; i < num_terms; i++) {
    printf("%d %f\n", i, host_parts[i]);
    total += host_parts[i];
   }
   printf("%f \n", 2*total);

   return 0;
}

