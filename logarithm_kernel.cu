#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

__global__ void logarithm(float *device_parts, float n, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < N) {
        float x = 2*idx + 1;
        device_parts[idx] = (1/x)*pow( (n - 1)/(n + 1), x);
    }
}


int num_terms = 128;
size_t size_terms = num_terms*sizeof(int);

int main() {
   int i;
   float z = 34.7;
   float *device_parts;
   hipMalloc((void**) &device_parts, size_terms);

   int ThreadsPerBlock = 16;
   int NumBlocks = (int) ((num_terms + ThreadsPerBlock - 1) / ThreadsPerBlock );
   logarithm <<< NumBlocks, ThreadsPerBlock, 1 >>> (device_parts, z, num_terms);
   float result = cublasSasum(num_terms, device_parts, 1);

   printf("%f \n", 2*result);

   hipFree(device_parts);

   return 0;
}

